#include "hip/hip_runtime.h"
// /opt/nvidia/hpc_sdk/Linux_x86_64/2022/compilers/bin/nvcc -ccbin =gcc-11 --gpu-code sm_75 --gpu-architecture compute_75 hello.cu -o hello && ./hello
// 
// x=0     x=4
// # . . # .  y=0
// #[. . .].
// #[# . .]#
// .[. # .].
// . . # # #  y=4

#include <stdio.h>
#include <numeric>
#include <vector>
#include <array>


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

int read(std::array<int, 512>&, std::vector<int>&, char*);

__device__ int idx_lookup(int x, int y) {
  int retval = x + COLS*y;
  if (0 <= retval && retval < COLS*ROWS) {
    return retval;
  } else {
    printf("computed index out of bound %d, %d -> %d\n", x, y, retval);
    return retval;
  }
}

__host__ int idx_lookup_h(int x, int y) {
  return x + COLS*y;
}

__device__ int input_idx_lookup(int* input, int x, int y) {
  if (x>=0 && x < COLS && y >=0 && y< ROWS) {
      return input[idx_lookup(x, y)];
  } else {
    // fingers crossed that the corner piece is "rest of the universe"
    return input[0];
  }
}

__global__ void iter(int* input_, int* output_, int* LUT_) {
  int x_cen = blockIdx.x;
  int y_cen = blockIdx.y;

  /* if (x_cen==0 || x_cen == COLS-1 || y_cen == 0 || y_cen == ROWS-1) { */
  /*   output_[idx_lookup(x_cen, y_cen)] = LUT_[0]; */
  /* } else { */
    int cur =
      (1<<8) * input_idx_lookup(input_, x_cen-1, y_cen-1) +
      (1<<7) * input_idx_lookup(input_, x_cen+0, y_cen-1) +
      (1<<6) * input_idx_lookup(input_, x_cen+1, y_cen-1) +

      (1<<5) * input_idx_lookup(input_, x_cen-1, y_cen+0) +
      (1<<4) * input_idx_lookup(input_, x_cen+0, y_cen+0) +
      (1<<3) * input_idx_lookup(input_, x_cen+1, y_cen+0) +

      (1<<2) * input_idx_lookup(input_, x_cen-1, y_cen+1) +
      (1<<1) * input_idx_lookup(input_, x_cen+0, y_cen+1) +
      (1<<0) * input_idx_lookup(input_, x_cen+1, y_cen+1);
    if (cur >= 512) {
      printf("1<<8 * %d\t 1<<7 * %d\t 1<<6 * %d\t 1<<5 * %d\t 1<<4 * %d\t 1<<3 * %d\t 1<<2 * %d\t 1<<1 * %d\t 1<<0 * %d = %d\n",
      input_idx_lookup(input_, x_cen-1, y_cen-1),
      input_idx_lookup(input_, x_cen+0, y_cen-1),
      input_idx_lookup(input_, x_cen+1, y_cen-1),

      input_idx_lookup(input_, x_cen-1, y_cen+0),
      input_idx_lookup(input_, x_cen+0, y_cen+0),
      input_idx_lookup(input_, x_cen+1, y_cen+0),

      input_idx_lookup(input_, x_cen-1, y_cen+1),
      input_idx_lookup(input_, x_cen+0, y_cen+1),
      input_idx_lookup(input_, x_cen+1, y_cen+1), cur);
    }
    if (LUT_[cur] != 0 && LUT_[cur] != 1) {
      printf("invalid LUT %d\n",LUT_[cur]);
    }
    output_[idx_lookup(x_cen, y_cen)] = LUT_[cur];
  /* } */
}

template <typename T>
void illustrate(T data) {
  for (std::size_t y = 0; y < COLS ; y++) {
    for (std::size_t x = 0; x < ROWS ; x++) {
      if (1==data[idx_lookup_h(x, y)]) {
        printf("#");
      } else if(0==data[idx_lookup_h(x, y)]) {
        printf(".");
      } else {
        printf("!!");
      }
    }
    printf("\n");
  }
}

template <typename T>
int val(T data) {
  int acc = 0;
  for (std::size_t y = 1; y < COLS-1 ; y++) {
    for (std::size_t x = 1; x < ROWS-1 ; x++) {
      acc += data[idx_lookup_h(x, y)];
    }
  }
  return acc;
}

template <std::size_t N>
void lutprint(const std::array<int, N> LUT) {
  for (auto c: LUT) {
    if (c==1) {
      printf("#");
    } else if (c==0) {
      printf(".");
    } else {
      printf("LUT BROKEN\n");
    }
  }
  printf("\n");
}

int main(int argc, char** argv) {
  int *input;
  int *intermediate_a;
  int *intermediate_b;
  int *output;
  int *LUT;
  hipMalloc((void**)&input, ROWS*COLS*sizeof(int));
  hipMalloc((void**)&intermediate_a, ROWS*COLS*sizeof(int));
  hipMalloc((void**)&intermediate_b, ROWS*COLS*sizeof(int));
  hipMalloc((void**)&output, ROWS*COLS*sizeof(int));
  hipMalloc((void**)&LUT, 512*sizeof(int));

  std::vector<int> host_image(ROWS*COLS, 0);
  std::array<int, 512> host_LUT;
  /* std::array<int, 512> invalid; */
  /* for (auto& e: invalid) { */
  /*   e = 0; */
  /* } */
  /* invalid[1<<4] = 1; */

  if (read(host_LUT, host_image, argv[1])) {
    printf("???\n");
    return 1;
  };
  lutprint(host_LUT);
  illustrate(host_image);

  HANDLE_ERROR(hipMemcpy(intermediate_a, host_image.data(), ROWS*COLS*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(LUT, host_LUT.data(), 512*sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipDeviceSynchronize());

  for (int i = 0; i < (PADDING-1)/2; ++i) {
    if (i%2 == 0) {
      iter<<<{ROWS, COLS, 1},1>>>(intermediate_a, intermediate_b, LUT);
    } else {
      iter<<<{ROWS, COLS, 1},1>>>(intermediate_b, intermediate_a, LUT);
    }
  }
  hipDeviceSynchronize();
  printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
  if (((PADDING-1)/2 - 1) == 0) {
    HANDLE_ERROR(hipMemcpy(host_image.data(), intermediate_b, ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost));
  } else {
    HANDLE_ERROR(hipMemcpy(host_image.data(), intermediate_a, ROWS*COLS*sizeof(int), hipMemcpyDeviceToHost));
  }

  illustrate(host_image);

  printf("result %d\n", val(host_image));

  hipFree(input);
  hipFree(intermediate_a);
  hipFree(intermediate_b);
  hipFree(output);
  hipFree(LUT);
  return 0;
}
